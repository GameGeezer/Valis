#include "TestScreen.cuh"

#include <map>
#include <glm/fwd.hpp>
#include <glm/glm.hpp>
#include <glm/mat4x4.hpp>
#include <glm/ext.hpp>

#include <iostream>

#include "hip/hip_runtime.h"

#include "SDFRenderer.cuh"
#include "PBO.cuh"
#include "Texture2D.cuh"
#include "CudaGLBufferMapping.cuh"
#include "Matrix4Device.cuh"
#include "ShaderProgram.cuh"
#include "Camera.cuh"
#include "Player.cuh"


void
TestScreen::onCreate()
{
	renderer = new SDFRenderer();
	pbo = new PBO(4 * 640 * 480);
	texture = new Texture2D(640, 480);
	mapping = new CudaGLBufferMapping(*pbo);
	windowBlockSize = new dim3(16, 16, 1);
	windowGridSize = new dim3(640 / windowBlockSize->x, 480 / windowBlockSize->y);

	Camera* camera = new Camera(640, 680, 0.1f, 1000.0f, 45.0f);
	camera->translate(0, 0, 1);

	player = new Player(*camera);
}

void
TestScreen::onPause()
{

}

void
TestScreen::onLeave()
{

}

void
TestScreen::onResume()
{

}

void
TestScreen::onUpdate(int delta)
{
	glm::mat4 invViewProjection;
	player->camera->constructInverseViewProjection(invViewProjection);
	renderer->renderToMapping(*mapping, *windowGridSize, *windowBlockSize, invViewProjection);
	pbo->bind();

	glDrawPixels(texture->width, texture->height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, texture->width, texture->height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
	pbo->unbind();
}

void
TestScreen::onResize(int width, int height)
{

}

void
TestScreen::onDestroy()
{

}