#include "hip/hip_runtime.h"
#include "SDFExtractor.cuh"

#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/sort.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"

#include "NumericBoolean.cuh"
#include "RenderPoint.cuh"
#include "SDFDevice.cuh"


__global__ void extractPointCloudAsBitArray(ExtractionBlock *d_output, SDFDevice *sdf, uint32_t clusterDim)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t z = blockIdx.z * blockDim.z + threadIdx.z;

	uint32_t gridDimension = (clusterDim * 4);

	if (x >= gridDimension || y >= gridDimension || z >= gridDimension)
	{
		return;
	}

	// The index of the cell in relation to 4 x 4 x 4 block of bits it's contained in
	uint32_t localX = x & 3;
	uint32_t localY = y & 3;
	uint32_t localZ = z & 3;

	uint32_t bitToFlip = localX + localY * 4 + localZ * 16;

	// Which cluster the cell is in
	uint32_t clusterX = x / 4;
	uint32_t clusterY = y / 4;
	uint32_t clusterZ = z / 4;

	uint32_t clusterIndex = clusterX + clusterY * clusterDim + clusterZ * clusterDim * clusterDim;

	float divisionsAsFloat = ((float)gridDimension);

	// normalized x, y, and z
	float normalizeX = ((float)x) / divisionsAsFloat;
	float normalizeY = ((float)y) / divisionsAsFloat;
	float normalizeZ = ((float)z) / divisionsAsFloat;

	// How far the cell is from the sdf
	float distance = sdf->distanceFromPoint(sdf->primitives, sdf->modifications, sdf->modificationCount, glm::vec3(normalizeX, normalizeY, normalizeZ));

	// Decide whether to generate a point
	float cellDimension = 1.0f / divisionsAsFloat;

	NumericBoolean shouldGeneratePoint = numericLessThan_float(distance, cellDimension) * numericGreaterThan_float(distance, 0);

	NumericBoolean writeFirst = numericLessThan_uint32_t(bitToFlip, 32);
	NumericBoolean writeSecond = numericNegate_uint32_t(writeFirst);

	bitToFlip = bitToFlip * writeFirst + (bitToFlip - 32) * writeSecond;

	uint32_t bitToOrWith = (1 << bitToFlip) *shouldGeneratePoint;
	uint32_t orFirst = bitToOrWith * writeFirst;
	uint32_t orSecond = bitToOrWith * writeSecond;
	atomicOr(&(d_output[clusterIndex].first), orFirst);
	atomicOr(&(d_output[clusterIndex].second), orSecond);
}

__global__ void createCloudFromBuffers(RenderPoint* d_output, ExtractionBlock *coverageBuffer, ExtractionBlock *materialBuffer, uint32_t subsectionClusterDim, uint32_t totalClusterDim, uint32_t clusterBufferSize, int dimensionOffsetX, int dimensionOffsetY, int dimensionOffsetZ)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t z = blockIdx.z * blockDim.z + threadIdx.z;

	// Check to see if x, y, or z exceeds the bounds of the local grid
	if (x >= subsectionClusterDim * 4 || y >= subsectionClusterDim * 4 || z >= subsectionClusterDim * 4)
	{
		return;
	}


	uint32_t offsetX = x + dimensionOffsetX;
	uint32_t offsetY = y + dimensionOffsetY;
	uint32_t offsetZ = z + dimensionOffsetZ;

	// Check to see if x, y, ot z exceed the bounds of the entire grid
	if (offsetX >= totalClusterDim * 4 || offsetY >= totalClusterDim * 4 || offsetZ >= totalClusterDim * 4)
	{
		return;
	}

	int outputIndex = x + y * totalClusterDim * 4 + z * totalClusterDim * totalClusterDim * 16;

	if (outputIndex >= clusterBufferSize)
	{
		return;
	}
	
	// x, y, and z relative to the cluster 0, 0 , 0
	uint32_t localX = x & 3;
	uint32_t localY = y & 3;
	uint32_t localZ = z & 3;

	uint32_t bitToCheck = localX + localY * 4 + localZ * 16;

	// Which cluster the cell is in
	uint32_t clusterX = offsetX / 4;
	uint32_t clusterY = offsetY / 4;
	uint32_t clusterZ = offsetZ / 4;

	// The cluster index relative to the entire grid
	uint32_t clusterIndex = clusterX + clusterY * totalClusterDim + clusterZ * totalClusterDim * totalClusterDim;

	ExtractionBlock surfaceCoverage = coverageBuffer[clusterIndex];
	ExtractionBlock materialCoverage = materialBuffer[clusterIndex];

	NumericBoolean checkFirst = numericLessThan_uint32_t(bitToCheck, 32);
	NumericBoolean checkSecond = numericNegate_uint32_t(checkFirst);

	bitToCheck = bitToCheck * checkFirst + (bitToCheck - 32) * checkSecond;

	uint32_t bitToAndWith = (1 << bitToCheck);

	uint32_t andCoverageFirst = (surfaceCoverage.first & materialCoverage.first) & bitToAndWith;
	uint32_t andCoverageSecond = (surfaceCoverage.second & materialCoverage.second) & bitToAndWith;

	NumericBoolean foundFirst = numericGreaterThan_uint32_t(andCoverageFirst * checkFirst, 0);
	NumericBoolean foundSecond = numericGreaterThan_uint32_t(andCoverageSecond * checkSecond, 0);

	NumericBoolean materialCoverageOverlap = numericGreaterThan_uint32_t(foundFirst + foundSecond, 0);

	uint32_t gridDimension = (totalClusterDim * 4);

	float divisionsAsFloat = ((float)gridDimension);

	float normalizeX = ((float)offsetX) / divisionsAsFloat;
	float normalizeY = ((float)offsetY) / divisionsAsFloat;
	float normalizeZ = ((float)offsetZ) / divisionsAsFloat;

	d_output[outputIndex].positionX = normalizeX * materialCoverageOverlap;
	d_output[outputIndex].positionY = normalizeY * materialCoverageOverlap;
	d_output[outputIndex].positionZ = normalizeZ * materialCoverageOverlap;
	
}


SDFExtractor::SDFExtractor(uint32_t clusterDensity, uint32_t extractionClusterDensity) : clusterDensity(clusterDensity), extractionClusterDensity(extractionClusterDensity)
{
	pointCoverageBuffer = new thrust::device_vector< ExtractionBlock >(clusterDensity * clusterDensity * clusterDensity);
	materialCoverageBuffer = new thrust::device_vector< ExtractionBlock >(clusterDensity * clusterDensity * clusterDensity);
	partialExtractionBuffer = new thrust::device_vector< RenderPoint >(extractionClusterDensity * extractionClusterDensity * extractionClusterDensity * 64);
}

struct is_not_zero
{
	__host__ __device__
	bool operator()(const RenderPoint& point)
	{
		return point.positionX != 0 && point.positionY != 0 && point.positionZ != 0;
	}
};

struct is_not_zero_extract
{
	__host__ __device__
		bool operator()(const ExtractionBlock& point)
	{
		return point.first != 0 && point.second != 0;
	}
};

struct is_not_zero_uint32_t
{
	__host__ __device__
	bool operator()(const uint32_t& point)
	{
		return point != 0;
	}
};

struct shiftRenderPointsLeft
{
	__host__ __device__
	bool operator()(const RenderPoint& point1, const RenderPoint& point2)
	{
		return (point1.positionX + point1.positionY + point1.positionZ) >  (point2.positionX + point2.positionY + point2.positionZ);
	}
};

thrust::host_vector< RenderPoint >*
SDFExtractor::extract(SDFDevice& sdf)
{

	thrust::host_vector< RenderPoint >* extractedPoints = new thrust::host_vector< RenderPoint >();

	thrust::fill(pointCoverageBuffer->begin(), pointCoverageBuffer->end(), ExtractionBlock());

	ExtractionBlock* pointCoverageStart = thrust::raw_pointer_cast(pointCoverageBuffer->data());

	dim3 blocksBitAr(clusterDensity / 2, clusterDensity / 2, clusterDensity / 2);
	dim3 threadsBitAr(8, 8, 8);
	extractPointCloudAsBitArray << <blocksBitAr, threadsBitAr >> >(pointCoverageStart, &sdf, clusterDensity);

	//thrust::host_vector< ExtractionBlock > checkExtract = *pointCoverageBuffer;
	//int count3 = thrust::count_if(checkExtract.begin(), checkExtract.end(), is_not_zero_extract());
	//int z = checkExtract.size();
	RenderPoint* partialExtractionStart = thrust::raw_pointer_cast(partialExtractionBuffer->data());

	dim3 partialExtractionBlocks(extractionClusterDensity / 2, extractionClusterDensity / 2, extractionClusterDensity / 2);
	dim3 partialExtractionThreads(8, 8, 8);
	int totalCreated = 0;
	for (int i = 0; i < clusterDensity; i += extractionClusterDensity)
	{
		for (int j = 0; j < clusterDensity; j += extractionClusterDensity)
		{
			for (int k = 0; k < clusterDensity; k += extractionClusterDensity)
			{
				createCloudFromBuffers << <partialExtractionBlocks, partialExtractionThreads >> > (partialExtractionStart, pointCoverageStart, pointCoverageStart, extractionClusterDensity, clusterDensity, partialExtractionBuffer->size(), i * 4, j * 4, k * 4);
				thrust::sort(partialExtractionBuffer->begin(), partialExtractionBuffer->end(), shiftRenderPointsLeft());
				thrust::host_vector< RenderPoint > checkExtract = *partialExtractionBuffer;
				int numberCreated = thrust::count_if(partialExtractionBuffer->begin(), partialExtractionBuffer->end(), is_not_zero());
				extractedPoints->resize(totalCreated + numberCreated);
				hipMemcpy(thrust::raw_pointer_cast(extractedPoints->data()) + totalCreated, partialExtractionStart, numberCreated * sizeof(RenderPoint), hipMemcpyDeviceToHost);
				totalCreated += numberCreated;
			}
		}
	}
	
	return extractedPoints;
}