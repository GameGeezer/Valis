#include "hip/hip_runtime.h"
#include "SDFExtractor.cuh"

#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/partition.h>
#include <thrust/execution_policy.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"

#include "NumericBoolean.cuh"
#include "RenderPoint.cuh"
#include "SDFDevice.cuh"

#include "CudaGLBufferMapping.cuh"

#include "VBO.cuh"
#include "PBO.cuh"


__global__ void extractVertexPlacementAsBitArray(ExtractionBlock *d_output, SDFDevice *sdf, uint32_t clusterDim)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t z = blockIdx.z * blockDim.z + threadIdx.z;

	uint32_t gridDimension = (clusterDim * 4);

	if (x > (gridDimension + 2) || y > (gridDimension + 2) || z > (gridDimension + 2))
	{
		return;
	}

	// The index of the cell in relation to 4 x 4 x 4 block of bits it's contained in
	uint32_t localX = x & 3;
	uint32_t localY = y & 3;
	uint32_t localZ = z & 3;

	uint32_t bitToFlip = localX + localY * 4 + localZ * 16;

	// Which cluster the cell is in
	uint32_t clusterX = x / 4;
	uint32_t clusterY = y / 4;
	uint32_t clusterZ = z / 4;

	uint32_t clusterIndex = clusterX + clusterY * clusterDim + clusterZ * clusterDim * clusterDim;

	float divisionsAsFloat = ((float)gridDimension);

	float offset = 1.0f / divisionsAsFloat;

	// normalized x, y, and z
	float normalizeX = (((float)x) / divisionsAsFloat) - offset;
	float normalizeY = (((float)y) / divisionsAsFloat) - offset;
	float normalizeZ = (((float)z) / divisionsAsFloat) - offset;

	// How far the cell is from the sdf
	float distance = sdf->distanceFromPoint(glm::vec3(normalizeX, normalizeY, normalizeZ));

	// Decide whether to generate a point
	float cellDimension = 1.0f / divisionsAsFloat;

	NumericBoolean isOutside = numericGreaterThan_float(distance, 0);

	NumericBoolean writeFirst = numericLessThan_uint32_t(bitToFlip, 32);
	NumericBoolean writeSecond = numericNegate_uint32_t(writeFirst);

	bitToFlip = bitToFlip * writeFirst + (bitToFlip - 32) * writeSecond;

	uint32_t bitToOrWith = (1 << bitToFlip) * isOutside;
	uint32_t orFirst = bitToOrWith * writeFirst;
	uint32_t orSecond = bitToOrWith * writeSecond;

	atomicOr(&(d_output[clusterIndex].first), orFirst);
	atomicOr(&(d_output[clusterIndex].second), orSecond);
}

__global__ void extractPointCloudAsBitArray(ExtractionBlock *d_output, SDFDevice *sdf, uint32_t clusterDim)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t z = blockIdx.z * blockDim.z + threadIdx.z;

	uint32_t gridDimension = (clusterDim * 4);

	if (x >= gridDimension || y >= gridDimension || z >= gridDimension)
	{
		return;
	}

	// The index of the cell in relation to 4 x 4 x 4 block of bits it's contained in
	uint32_t localX = x & 3;
	uint32_t localY = y & 3;
	uint32_t localZ = z & 3;

	uint32_t bitToFlip = localX + localY * 4 + localZ * 16;

	// Which cluster the cell is in
	uint32_t clusterX = x / 4;
	uint32_t clusterY = y / 4;
	uint32_t clusterZ = z / 4;

	uint32_t clusterIndex = clusterX + clusterY * clusterDim + clusterZ * clusterDim * clusterDim;

	float divisionsAsFloat = ((float)gridDimension);

	// normalized x, y, and z
	float normalizeX = ((float)x) / divisionsAsFloat;
	float normalizeY = ((float)y) / divisionsAsFloat;
	float normalizeZ = ((float)z) / divisionsAsFloat;

	// How far the cell is from the sdf
	float distance = sdf->distanceFromPoint(glm::vec3(normalizeX, normalizeY, normalizeZ));

	// Decide whether to generate a point
	float cellDimension = 1.0f / divisionsAsFloat;

	NumericBoolean shouldGeneratePoint = numericLessThan_float(distance, cellDimension) * numericGreaterThan_float(distance, 0);

	NumericBoolean writeFirst = numericLessThan_uint32_t(bitToFlip, 32);
	NumericBoolean writeSecond = numericNegate_uint32_t(writeFirst);

	bitToFlip = bitToFlip * writeFirst + (bitToFlip - 32) * writeSecond;

	uint32_t bitToOrWith = (1 << bitToFlip) *shouldGeneratePoint;
	uint32_t orFirst = bitToOrWith * writeFirst;
	uint32_t orSecond = bitToOrWith * writeSecond;

	atomicOr(&(d_output[clusterIndex].first), orFirst);
	atomicOr(&(d_output[clusterIndex].second), orSecond);
}

__device__ __inline__ NumericBoolean
isExtractionBlockBitFlipped(ExtractionBlock *buffer, uint32_t x, uint32_t y, uint32_t z, uint32_t offsetX, uint32_t offsetY, uint32_t offsetZ, uint32_t subsectionClusterDim, uint32_t totalClusterDim)
{
	uint32_t localX = x & 3;
	uint32_t localY = y & 3;
	uint32_t localZ = z & 3;

	uint32_t bitToCheck = localX + localY * 4 + localZ * 16;

	// Which cluster the cell is in
	uint32_t clusterX = offsetX / 4;
	uint32_t clusterY = offsetY / 4;
	uint32_t clusterZ = offsetZ / 4;

	// The cluster index relative to the entire grid
	uint32_t clusterIndex = clusterX + clusterY * totalClusterDim + clusterZ * totalClusterDim * totalClusterDim;

	NumericBoolean checkFirst = numericLessThan_uint32_t(bitToCheck, 32);
	NumericBoolean checkSecond = numericNegate_uint32_t(checkFirst);

	bitToCheck = bitToCheck * checkFirst + (bitToCheck - 32) * checkSecond;

	uint32_t bitToAndWith = (1 << bitToCheck);

	ExtractionBlock bufferValue = buffer[clusterIndex];

	uint32_t andCoverageFirst = bufferValue.first & bitToAndWith;
	uint32_t andCoverageSecond = bufferValue.second & bitToAndWith;

	NumericBoolean foundFirst = numericGreaterThan_uint32_t(andCoverageFirst * checkFirst, 0);
	NumericBoolean foundSecond = numericGreaterThan_uint32_t(andCoverageSecond * checkSecond, 0);

	return foundFirst + foundSecond;
}

__global__ void createCloudFromBuffers(RenderPoint* d_output, ExtractionBlock * gridIntersection, ExtractionBlock *coverageBuffer, ExtractionBlock *materialBuffer, uint32_t subsectionClusterDim, uint32_t totalClusterDim, uint32_t clusterBufferSize, int dimensionOffsetX, int dimensionOffsetY, int dimensionOffsetZ)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t z = blockIdx.z * blockDim.z + threadIdx.z;

	// Check to see if x, y, or z exceeds the bounds of the local grid
	uint32_t subGridDimension = (subsectionClusterDim * 4);
	if (x >= subGridDimension || y >= subGridDimension || z >= subGridDimension)
	{
		return;
	}

	uint32_t offsetX = x + dimensionOffsetX;
	uint32_t offsetY = y + dimensionOffsetY;
	uint32_t offsetZ = z + dimensionOffsetZ;

	// Check to see if x, y, ot z exceed the bounds of the entire grid
	uint32_t gridDimension = (totalClusterDim * 4);
	if (offsetX >= gridDimension || offsetY >= gridDimension || offsetZ >= gridDimension)
	{
		return;
	}

	// Make sure we don't write out of bounds
	int outputIndex = x + y * subGridDimension + z * subGridDimension * subGridDimension;
	if (outputIndex >= clusterBufferSize)
	{
		return;// Ideally this should never be hit, look into what it is.
	}


	NumericBoolean coverageFlipped = isExtractionBlockBitFlipped(coverageBuffer, x, y, z, offsetX, offsetY, offsetZ, subsectionClusterDim, totalClusterDim);
	/*
	NumericBoolean bottomLeftBack = isExtractionBlockBitFlipped(gridIntersection, x, y, z, offsetX, offsetY, offsetZ, subsectionClusterDim, totalClusterDim);
	NumericBoolean bottomRightBack = isExtractionBlockBitFlipped(gridIntersection, x + 1, y, z, offsetX + 1, offsetY, offsetZ, subsectionClusterDim, totalClusterDim);
	NumericBoolean topLeftBack = isExtractionBlockBitFlipped(gridIntersection, x, y + 1, z, offsetX, offsetY + 1, offsetZ, subsectionClusterDim, totalClusterDim);
	NumericBoolean bottomLeftForward = isExtractionBlockBitFlipped(gridIntersection, x, y, z + 1, offsetX, offsetY, offsetZ + 1, subsectionClusterDim, totalClusterDim);
	NumericBoolean topRightBack = isExtractionBlockBitFlipped(gridIntersection, x + 1, y + 1, z, offsetX + 1, offsetY + 1, offsetZ, subsectionClusterDim, totalClusterDim);
	NumericBoolean BottmRightForward = isExtractionBlockBitFlipped(gridIntersection, x + 1, y, z + 1, offsetX + 1, offsetY, offsetZ + 1, subsectionClusterDim, totalClusterDim);
	NumericBoolean TopLeftForward = isExtractionBlockBitFlipped(gridIntersection, x, y + 1, z + 1, offsetX, offsetY + 1, offsetZ + 1, subsectionClusterDim, totalClusterDim);
	NumericBoolean TopRightForward = isExtractionBlockBitFlipped(gridIntersection, x + 1, y + 1, z + 1, offsetX + 1, offsetY + 1, offsetZ + 1, subsectionClusterDim, totalClusterDim);
	*/
	//NumericBoolean materialFlipped = isExtractionBlockBitFlipped(coverageBuffer, x, y, z, offsetX, offsetY, offsetZ, subsectionClusterDim, totalClusterDim);

	NumericBoolean materialCoverageOverlap = numericGreaterThan_uint32_t(coverageFlipped, 0);

	float divisionsAsFloat = ((float)gridDimension);

	float normalizeX = ((float)offsetX) / divisionsAsFloat;
	float normalizeY = ((float)offsetY) / divisionsAsFloat;
	float normalizeZ = ((float)offsetZ) / divisionsAsFloat;

	d_output[outputIndex].positionX = normalizeX * materialCoverageOverlap;
	d_output[outputIndex].positionY = normalizeY * materialCoverageOverlap;
	d_output[outputIndex].positionZ = normalizeZ * materialCoverageOverlap;
}

__global__ void copyLocal(RenderPoint* d_output, RenderPoint *coverageBuffer, uint32_t blockWidth, uint32_t pointsToCopy)
{
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	uint32_t index = x + y * blockWidth;

	if (x >= pointsToCopy)
	{
		return;
	}

	d_output[index] = coverageBuffer[index];
}


SDFExtractor::SDFExtractor(uint32_t clusterDensity, uint32_t extractionClusterDensity) : 
	clusterDensity(clusterDensity), 
	extractionClusterDensity(extractionClusterDensity), 
	coverageExtractBlockDim(clusterDensity / 2, clusterDensity / 2, clusterDensity / 2),
	partialExtractionBlockDim(extractionClusterDensity / 2, extractionClusterDensity / 2, extractionClusterDensity / 2),
	parseThreadsDim(8, 8, 8),
	pointCoverageBuffer(new thrust::device_vector< ExtractionBlock >(clusterDensity * clusterDensity * clusterDensity)),
	materialCoverageBuffer(new thrust::device_vector< ExtractionBlock >(clusterDensity * clusterDensity * clusterDensity)),
	partialExtractionBuffer(new thrust::device_vector< RenderPoint >(extractionClusterDensity * extractionClusterDensity * extractionClusterDensity * 64)),
	sdfGridIntersectionBuffer(new thrust::device_vector< ExtractionBlock >((clusterDensity + 2) * (clusterDensity + 2) * (clusterDensity + 2)))
{

}

SDFExtractor::~SDFExtractor()
{
	delete pointCoverageBuffer;
	delete materialCoverageBuffer;
	delete partialExtractionBuffer;
}

struct is_not_zero
{
	__host__ __device__
	bool operator()(const RenderPoint& point)
	{
		return point.positionX + point.positionY + point.positionZ != 0;
	}
};

struct is_not_zero_extract
{
	__host__ __device__
		bool operator()(const ExtractionBlock& point)
	{
		return point.first != 0 && point.second != 0;
	}
};

struct is_not_zero_uint32_t
{
	__host__ __device__
	bool operator()(const uint32_t& point)
	{
		return point != 0;
	}
};

struct shiftRenderPointsLeft
{
	__host__ __device__
	bool operator()(const RenderPoint& point1, const RenderPoint& point2)
	{
		return (point1.positionX + point1.positionY + point1.positionZ) >  (point2.positionX + point2.positionY + point2.positionZ);
	}
};

thrust::host_vector< RenderPoint >*
SDFExtractor::extract(SDFDevice& sdf)
{
	// Zero the coverage buffer
	thrust::fill(pointCoverageBuffer->begin(), pointCoverageBuffer->end(), ExtractionBlock());
	// Point to the coverage buffer
	ExtractionBlock* pointCoverageStart = thrust::raw_pointer_cast(pointCoverageBuffer->data());
	// Extract the coverage buffer
	//extractPointCloudAsBitArray << <coverageExtractBlockDim, parseThreadsDim >> >(pointCoverageStart, &sdf, clusterDensity);
	// Point to the partial extraction buffer
	RenderPoint* partialExtractionStart = thrust::raw_pointer_cast(partialExtractionBuffer->data());
	// Create the buffer where all points will be stored
	thrust::host_vector< RenderPoint >* extractedPoints = new thrust::host_vector< RenderPoint >();
	// How many points have been created thus far
	int totalCreated = 0;
	for (int i = 0; i < clusterDensity; i += extractionClusterDensity)
	{
		for (int j = 0; j < clusterDensity; j += extractionClusterDensity)
		{
			for (int k = 0; k < clusterDensity; k += extractionClusterDensity)
			{
				//createCloudFromBuffers << <partialExtractionBlockDim, parseThreadsDim >> > (partialExtractionStart, pointCoverageStart, pointCoverageStart, extractionClusterDensity, clusterDensity, partialExtractionBuffer->size(), i * 4, j * 4, k * 4);
				thrust::sort(partialExtractionBuffer->begin(), partialExtractionBuffer->end(), shiftRenderPointsLeft());
				thrust::host_vector< RenderPoint > checkExtract = *partialExtractionBuffer;
				int numberCreated = thrust::count_if(partialExtractionBuffer->begin(), partialExtractionBuffer->end(), is_not_zero());
				extractedPoints->resize(totalCreated + numberCreated);
				hipMemcpy(thrust::raw_pointer_cast(extractedPoints->data()) + totalCreated, partialExtractionStart, numberCreated * sizeof(RenderPoint), hipMemcpyDeviceToHost);
				totalCreated += numberCreated;
			}
		}
	}
	
	return extractedPoints;
}

size_t
SDFExtractor::extractDynamic(SDFDevice& sdf, CudaGLBufferMapping<RenderPoint>& mapping)
{
	mapping.map();
	size_t bufferLength = mapping.getSizeInBytes() / sizeof(RenderPoint);
	thrust::device_ptr<RenderPoint> bufferPointer = thrust::device_pointer_cast(mapping.getDeviceOutput());

	// Zero the coverage buffer
	thrust::fill(sdfGridIntersectionBuffer->begin(), sdfGridIntersectionBuffer->end(), ExtractionBlock());
	thrust::fill(pointCoverageBuffer->begin(), pointCoverageBuffer->end(), ExtractionBlock());
	// Point to the coverage buffer
	ExtractionBlock* gridIntersectionRaw = thrust::raw_pointer_cast(sdfGridIntersectionBuffer->data());
	ExtractionBlock* pointCoverageRaw = thrust::raw_pointer_cast(pointCoverageBuffer->data());
	// Extract the coverage buffer
	//extractVertexPlacementAsBitArray << <coverageExtractBlockDim, parseThreadsDim >> >(gridIntersectionRaw, &sdf, clusterDensity);
	extractPointCloudAsBitArray << <coverageExtractBlockDim, parseThreadsDim >> >(pointCoverageRaw, &sdf, clusterDensity);
	// Point to the partial extraction buffer
	RenderPoint* partialExtractionRaw = thrust::raw_pointer_cast(partialExtractionBuffer->data());

	// How many points have been created thus far
	size_t totalCreated = 0;
	for (int i = 0; i < clusterDensity; i += extractionClusterDensity)
	{
		for (int j = 0; j < clusterDensity; j += extractionClusterDensity)
		{
			for (int k = 0; k < clusterDensity; k += extractionClusterDensity)
			{
				createCloudFromBuffers << <partialExtractionBlockDim, parseThreadsDim >> > (partialExtractionRaw, gridIntersectionRaw, pointCoverageRaw, pointCoverageRaw, extractionClusterDensity, clusterDensity, partialExtractionBuffer->size(), i * 4, j * 4, k * 4);
				
				//Improve performance by eliminating this copy to the CPU
				int numberCreated = thrust::count_if(partialExtractionBuffer->begin(), partialExtractionBuffer->end(), is_not_zero());

				thrust::copy_if(partialExtractionBuffer->begin(), partialExtractionBuffer->end(), bufferPointer + totalCreated, is_not_zero());
				
				totalCreated += numberCreated;
			}
		}
	}

	mapping.unmap();

	return totalCreated;
}

size_t
SDFExtractor::extractRelative(SDFDevice& sdf, CudaGLBufferMapping<RenderPoint>& mapping, PBO& pbo)
{
	mapping.map();
	size_t bufferLength = mapping.getSizeInBytes() / sizeof(RenderPoint);
	thrust::device_ptr<RenderPoint> bufferPointer = thrust::device_pointer_cast(mapping.getDeviceOutput());

	// Zero the coverage buffer
	thrust::fill(pointCoverageBuffer->begin(), pointCoverageBuffer->end(), ExtractionBlock());
	// Point to the coverage buffer
	ExtractionBlock* pointCoverageRaw = thrust::raw_pointer_cast(pointCoverageBuffer->data());
	// Extract the coverage buffer
	//extractPointCloudAsBitArray << <coverageExtractBlockDim, parseThreadsDim >> >(pointCoverageRaw, &sdf, clusterDensity);
	// Point to the partial extraction buffer
	RenderPoint* partialExtractionRaw = thrust::raw_pointer_cast(partialExtractionBuffer->data());

	// How many points have been created thus far
	size_t totalCreated = 0;
	for (int i = 0; i < clusterDensity; i += extractionClusterDensity)
	{
		for (int j = 0; j < clusterDensity; j += extractionClusterDensity)
		{
			for (int k = 0; k < clusterDensity; k += extractionClusterDensity)
			{
				//createCloudFromBuffers << <partialExtractionBlockDim, parseThreadsDim >> > (partialExtractionRaw, pointCoverageRaw, pointCoverageRaw, extractionClusterDensity, clusterDensity, partialExtractionBuffer->size(), i * 4, j * 4, k * 4);

				//Improve performance by eliminating this copy to the CPU
				int numberCreated = thrust::count_if(partialExtractionBuffer->begin(), partialExtractionBuffer->end(), is_not_zero());

				thrust::copy_if(partialExtractionBuffer->begin(), partialExtractionBuffer->end(), bufferPointer + totalCreated, is_not_zero());

				totalCreated += numberCreated;
			}
		}
	}

	mapping.unmap();



	return totalCreated;
}